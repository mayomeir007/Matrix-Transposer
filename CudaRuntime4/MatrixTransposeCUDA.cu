#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include "MatrixTransposeCUDA.cuh"

using namespace std::chrono;

__global__ void warm_up_kernel(int* mat, int* transpose, int nx, int ny)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        transpose[ix * ny + iy] = mat[iy * nx + ix];
    }
}

__global__ void transpose_unroll4_col(int* mat, int* transpose, int ny, int nx)
{
    int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int ti = iy * nx + ix;
    int to = ix * ny + iy;

    if (ix < nx && iy < ny)
    {
        transpose[ti] = mat[to];
    }

    if (ix + blockDim.x < nx && iy < ny)
    {
        transpose[ti + blockDim.x] = mat[to + blockDim.x * ny];
    }

    if (ix + 2 * blockDim.x < nx && iy < ny)
    {
        transpose[ti + 2 * blockDim.x] = mat[to + 2 * blockDim.x * ny];
    }

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        transpose[ti + 3 * blockDim.x] = mat[to + 3 * blockDim.x * ny];
    }
}

bool MatrixTransposeCUDA(int nx, int ny)
{
    if (nx < 1 || nx > 20000 || ny < 1 || ny > 20000)
    {
        return false;
    }

    int block_x = 128;
    int block_y = 8;

    int size = nx * ny;
    int byte_size = sizeof(int) * size;

    printf("Matrix transpose for %d X %d matrix \n", nx, ny);

    int* h_mat_array = (int*)malloc(byte_size);
    int* h_trans_array = (int*)malloc(byte_size);
    int* h_ref = (int*)malloc(byte_size);

    //initialize matrix with integers between 0 and 255
    initialize(h_mat_array, size);
    //matirx transpose in CPU
    //clock_t cpu_start, cpu_end;
    //cpu_start = clock();
    auto cpu_start = high_resolution_clock::now();
    mat_transpose_cpu(h_mat_array, h_trans_array, nx, ny);
    auto cpu_end = high_resolution_clock::now();
    
    auto duration = duration_cast<microseconds>(cpu_end - cpu_start);
    //cpu_end = clock();

    //printf("CPU execution time : %4.6f \n",(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

    int* d_mat_array, * d_trans_array;

    hipMalloc((void**)&d_mat_array, byte_size);
    hipMalloc((void**)&d_trans_array, byte_size);

    dim3 blocks(block_x, block_y);

    unsigned int transGridDimX = (ny + block_x - 1) / block_x;
    unsigned int transGridDimY = (nx + block_y - 1) / block_y;
    dim3 grid_col_unroll((transGridDimX + 3) / 4, transGridDimY);
    /////warm up /////////////////////////////////////
    //hipMemcpy(d_mat_array, h_mat_array, byte_size, hipMemcpyHostToDevice);

    //transpose_unroll4_col << < grid_col_unroll, blocks >> > (d_mat_array, d_trans_array, nx, ny);

    //hipDeviceSynchronize();

    ////copy the transpose memroy back to cpu
    //hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost);
    //hipMemset(d_trans_array, 0, byte_size);
    ///////////////////////////////////
    hipEvent_t start, end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    hipMemcpy(d_mat_array, h_mat_array, byte_size, hipMemcpyHostToDevice);

    transpose_unroll4_col << < grid_col_unroll, blocks >> > (d_mat_array, d_trans_array, nx, ny);

    hipDeviceSynchronize();

    //copy the transpose memroy back to cpu
    hipMemcpy(h_ref, d_trans_array, byte_size, hipMemcpyDeviceToHost);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float time;
    hipEventElapsedTime(&time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

    //compare the CPU and GPU transpose matrix for validity
    compare_arrays(h_ref, h_trans_array, size);

    printf("CPU execution time : %f seconds\n", duration.count()/1000000.f);

    printf("GPU execution time : %f seconds\n", time/1000.f);

    float gpu_speed_factor = (duration.count() / time) / 1000.f;
    printf("GPU executed the matrix transpose %f times faster. \n", gpu_speed_factor);
    if (gpu_speed_factor > 2)
    {
        printf("Doing more operations on to the GPU would be a good idea.\n");
    }
    else
    {
        printf("Sorry the capability of the GPU didn't impress you. \nIf you havent ran this program in a while, consider running it again a few more times to warm up the GPU.\n");
    }
    hipDeviceReset();
    return true;
}

void initialize(int* input, const int array_size)
{
    srand(time(0));
    for (int i = 0; i < array_size; i++)
    {
        input[i] = (int)(rand() & 0xFF);
    }
}

void mat_transpose_cpu(int* mat, int* transpose, int nx, int ny)
{
    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            transpose[ix * ny + iy] = mat[iy * nx + ix];
        }
    }
}

void compare_arrays(int* a, int* b, int size)
{
    for (int i = 0; i < size; i++)
    {
        if (a[i] != b[i])
        {
            printf("Array transpose using CPU and array transpose using GPU are different! \n");
            printf("at index %d - CPU value %d | value GPU %d \n", i, a[i], b[i]);
            return;
        }
    }
    printf("Array transposed using CPU and array transposed using GPU are the same :) \n");
}