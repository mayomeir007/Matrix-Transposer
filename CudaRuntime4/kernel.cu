﻿#include <stdlib.h>
#include <stdio.h>

#include "MatrixTransposeCUDA.cuh"


int main(int argc, char** argv)
{
    bool valid = false;
    if(argc > 1)
    {
        valid = MatrixTransposeCUDA(atoi(argv[1]), atoi(argv[2]));
    }
    if (!valid)
    {
        printf("Invalid input\n");
    }
    return 0;
}
